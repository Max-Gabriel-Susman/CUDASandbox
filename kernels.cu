
#include <hip/hip_runtime.h>
#include <stdio.h> 

// a kernel is defined using the __global__ declaration specifier   
__global__ void add(int* a, int* b, int* c) {
    // each thread that executes the kernel is given a unique thread ID, 
    // this thread ID is accessible within the kernel through built-in variables(e.g. threadIdx, blockDim, & blockIdx). 
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

__managed__ int vector_a[256], vector_b[256], vector_c[256];

int main() {
    for (int i = 0; i < 256; i++) {
        vector_a[i] = i;

        vector_b[i] = 256 - i; 
    }

    // the number of CUDA threads that execute that kernal for a give kernel call, 
    // is specified using a new <<<...>>> execution configuration syntax 
    // in the execution configuration syntax used below the 'add' kernel is executed 
    // 256 times by 256 different CUDA threads as opposed to only once like a 
    // regular c++ function.
    add<<<1, 256>>>(vector_a, vector_b, vector_c);

    hipDeviceSynchronize();

    int result_sum = 0; 

    for (int i = 0; i < 256; i++) {
        result_sum += vector_c[i];
    }

    printf("Result: sum = % d", result_sum);
    return 0; 
}